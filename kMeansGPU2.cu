#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "kMeansGPU2.cuh"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <math.h>

#define HIP_INF_F 0x7ff0000000000000

#define CDR_ERR "hipDeviceReset failed!"
#define CSD_ERR "cudaDeviceSet failed!"
#define CGLE_ERR "Kernel launch failed!"
#define CDS_ERR "hipDeviceSynchronize returned error code"

template <int numberOfDimensions> void KMeansGPU2Solver<numberOfDimensions>::cudaCheck(hipError_t status, char* message) {
	if (status == hipSuccess)
		return;
	fprintf(stderr, "%s", message);
	exit(1);
}

struct is_equal_to
{
	is_equal_to(int number) {
		this->number = number;
	}

	__host__ __device__
		bool operator()(int x)
	{
		return x == number;
	}
	int number;
};

template <int numberOfDimensions> __device__ int findNearestClusterFor2(float* vector, float* centroidVectors, int centroidVectorLength)
{
	int minDistanceIndex = 0;
	float minDistanceSquared = HIP_INF_F;
	float distanceSquared = 0;

	for (int i = 0; i < centroidVectorLength; i++) {
		distanceSquared = 0;
		for (int j = 0; j < numberOfDimensions; j++)
			distanceSquared += powf(vector[j] - centroidVectors[i * numberOfDimensions + j], 2);
		if (distanceSquared < minDistanceSquared) {
			minDistanceSquared = distanceSquared;
			minDistanceIndex = i;
		}
	}

	return minDistanceIndex;
}

template <int numberOfDimensions> __global__ void findNearestClustersKernel2(float* dataVectors, int* centroidMemberships, int* membershipChangeVector, float* centroidVectors, int centroidVectorLength, int dataVectorsLength)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= dataVectorsLength)
		return;

	int index = findNearestClusterFor2<numberOfDimensions>(&(dataVectors[tid * numberOfDimensions]), centroidVectors, centroidVectorLength);
	if (centroidMemberships[tid * numberOfDimensions] != index * numberOfDimensions) {
		membershipChangeVector[tid] = 1;
		for (int i = 0; i < numberOfDimensions; i++)
			centroidMemberships[tid * numberOfDimensions + i] = index * numberOfDimensions + i;
	}	
}

template <int numberOfDimensions> void KMeansGPU2Solver<numberOfDimensions>::solve()
{
	float membershipChangeFraction;
	int iteration = 0;

	do {
		iteration++;
		clearVariables();
		findNearestClusters();
		averageNewClusters();
	} while (iteration < limit && getMembershipChangeFraction() > threshold);

}


template <int numberOfDimensions> void KMeansGPU2Solver<numberOfDimensions>::initSolver(float* dataVectors, int dataVectorLength, int centroidVectorLength, float threshold, int limit)
{
	cudaCheck(hipSetDevice(0), CSD_ERR);

	hipMallocManaged(&(this->dataVectors), dataVectorLength * numberOfDimensions * sizeof(float));
	thrust::copy(dataVectors, dataVectors + dataVectorLength * numberOfDimensions, this->dataVectors);
		

	this->dataVectorLength = dataVectorLength;
	this->centroidVectorLength = centroidVectorLength;
	this->threshold = threshold;
	this->limit = limit;

	copiedData = new float[dataVectorLength * numberOfDimensions];
	copiedMemberships = new float[dataVectorLength * numberOfDimensions];
	centroids = new float[centroidVectorLength * numberOfDimensions];
	keys = new int[centroidVectorLength * numberOfDimensions];

	hipMallocManaged(&membershipChangeVector, dataVectorLength * sizeof(int));
	hipMallocManaged(&centroidMemberships, dataVectorLength * numberOfDimensions * sizeof(int));
	thrust::fill(centroidMemberships, centroidMemberships + dataVectorLength * numberOfDimensions, -1);

	hipMallocManaged(&centroidVectors, centroidVectorLength * numberOfDimensions  * sizeof(float));

	hipMallocManaged(&centroidKeys, centroidVectorLength * sizeof(int));

	for (int i = 0; i < centroidVectorLength; i++)
		for (int j = 0; j < numberOfDimensions; j++)
			centroidVectors[i * numberOfDimensions + j] = (this->dataVectors)[i * numberOfDimensions + j];

}

template <int numberOfDimensions> void KMeansGPU2Solver<numberOfDimensions>::clearVariables()
{
	thrust::fill(membershipChangeVector, membershipChangeVector + dataVectorLength, 0);
}

template <int numberOfDimensions> float KMeansGPU2Solver<numberOfDimensions>::getMembershipChangeFraction()
{
	int membershipChangeCounter = thrust::reduce(membershipChangeVector, membershipChangeVector + dataVectorLength);
	return (float)membershipChangeCounter / dataVectorLength;
}

template <int numberOfDimensions> void KMeansGPU2Solver<numberOfDimensions>::findNearestClusters()
{
	int blockSize = 1024;
	int blocks = dataVectorLength / 1024 + 1;
	findNearestClustersKernel2<numberOfDimensions><<<blocks, blockSize>>>(dataVectors, centroidMemberships, membershipChangeVector, centroidVectors, centroidVectorLength, dataVectorLength);
	cudaCheck(hipGetLastError(), CGLE_ERR);
	cudaCheck(hipDeviceSynchronize(), CDS_ERR);
}

template <int numberOfDimensions> void KMeansGPU2Solver<numberOfDimensions>::averageNewClusters()
{
	thrust::copy(dataVectors, dataVectors + dataVectorLength * numberOfDimensions, copiedData);
	thrust::copy(centroidMemberships, centroidMemberships + dataVectorLength * numberOfDimensions, copiedMemberships);
	thrust::sort_by_key(copiedMemberships, copiedMemberships + dataVectorLength * numberOfDimensions, copiedData);

	thrust::reduce_by_key(
		copiedMemberships,
		copiedMemberships + dataVectorLength * numberOfDimensions,
		copiedData,
		keys,
		centroids
	);

	for (int i = 0; i < centroidVectorLength; i++) {
		int centroidMembershipCount = thrust::count_if(copiedMemberships, copiedMemberships + dataVectorLength * numberOfDimensions, is_equal_to(i*numberOfDimensions));
		for (int j = 0; j < numberOfDimensions; j++)
			centroidVectors[i * numberOfDimensions + j] = centroids[i * numberOfDimensions + j] / centroidMembershipCount;
	}

}

template <int numberOfDimensions> void KMeansGPU2Solver<numberOfDimensions>::clearSolver()
{
	cudaCheck(hipDeviceReset(), CDR_ERR);
}
