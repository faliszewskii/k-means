#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "kMeansGPU1.cuh"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <math.h>

#define HIP_INF_F 0x7ff0000000000000

#define CDR_ERR "hipDeviceReset failed!"
#define CSD_ERR "cudaDeviceSet failed!"
#define CGLE_ERR "Kernel launch failed!"
#define CDS_ERR "hipDeviceSynchronize returned error code"

template <int numberOfDimensions> void KMeansGPU1Solver<numberOfDimensions>::cudaCheck(hipError_t status, char* message) {
	if (status == hipSuccess)
		return;
	fprintf(stderr, "%s", message);
	exit(1);
}

template <int numberOfDimensions> __device__ int findNearestClusterFor(float* vector, float* centroidVectors, int centroidVectorLength)
{
	int minDistanceIndex = 0;
	float minDistanceSquared = HIP_INF_F;
	float distanceSquared = 0;

	for (int i = 0; i < centroidVectorLength; i++) {
		distanceSquared = 0;
		for (int j = 0; j < numberOfDimensions; j++)
			distanceSquared += powf(vector[j] - centroidVectors[i * numberOfDimensions + j], 2);
		if (distanceSquared < minDistanceSquared) {
			minDistanceSquared = distanceSquared;
			minDistanceIndex = i;
		}
	}

	return minDistanceIndex;
}

template <int numberOfDimensions> __global__ void findNearestClustersKernel(float* dataVectors, int* centroidMemberships, int* membershipChangeCounter, float* newCentroidVectors, int* centroidMembershipCounts, float* centroidVectors, int centroidVectorLength, int dataVectorsLength)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= dataVectorsLength)
		return;

	int index = findNearestClusterFor<numberOfDimensions>(&dataVectors[tid * numberOfDimensions], centroidVectors, centroidVectorLength);
	if (centroidMemberships[tid] != index) {
		atomicAdd(membershipChangeCounter, 1);
		centroidMemberships[tid] = index;
	}
	for (int j = 0; j < numberOfDimensions; j++)
		atomicAdd(&newCentroidVectors[index * numberOfDimensions + j], dataVectors[tid * numberOfDimensions + j]);
	atomicAdd(&centroidMembershipCounts[index], 1);
}

template <int numberOfDimensions> void KMeansGPU1Solver<numberOfDimensions>::solve()
{
	float membershipChangeFraction;
	int iteration = 0;

	do {
		clearVariables();
		findNearestClusters();
		averageNewClusters();
		membershipChangeFraction = (float)*membershipChangeCounter / dataVectorLength;
	} while (iteration < limit && membershipChangeFraction > threshold);

}


template <int numberOfDimensions> void KMeansGPU1Solver<numberOfDimensions>::initSolver(float* dataVectors, int dataVectorLength, int centroidVectorLength, float threshold, int limit)
{
	cudaCheck(hipSetDevice(0), CSD_ERR);

	hipMallocManaged(&(this->dataVectors), dataVectorLength * numberOfDimensions * sizeof(float));
	for (int i = 0; i < dataVectorLength * numberOfDimensions; i++)
		this->dataVectors[i] = dataVectors[i];

	this->dataVectorLength = dataVectorLength;
	this->centroidVectorLength = centroidVectorLength;
	this->threshold = threshold;
	this->limit = limit;

	hipMallocManaged(&membershipChangeCounter, sizeof(int));
	*(membershipChangeCounter) = 0;

	hipMallocManaged(&(centroidMemberships), dataVectorLength  * sizeof(int));
	hipMallocManaged(&(centroidMembershipCounts), centroidVectorLength * sizeof(int));
	
	hipMallocManaged(&(centroidVectors), centroidVectorLength * numberOfDimensions * sizeof(float));
	hipMallocManaged(&(newCentroidVectors), centroidVectorLength * numberOfDimensions * sizeof(float));

	for (int i = 0; i < centroidVectorLength; i++)
		for (int j = 0; j < numberOfDimensions; j++)
			centroidVectors[i * numberOfDimensions + j] = dataVectors[i * numberOfDimensions + j];

}

template <int numberOfDimensions> void KMeansGPU1Solver<numberOfDimensions>::clearVariables()
{
	*membershipChangeCounter = 0;

	for (int i = 0; i < centroidVectorLength; i++)
		for (int j = 0; j < numberOfDimensions; j++)
			newCentroidVectors[i * numberOfDimensions + j] = 0;

	for (int i = 0; i < centroidVectorLength; i++)
		centroidMembershipCounts[i] = 0;
}

template <int numberOfDimensions> void KMeansGPU1Solver<numberOfDimensions>::findNearestClusters()
{
	int blockSize = 1024;
	int blocks = dataVectorLength / 1024 + 1;
	findNearestClustersKernel<numberOfDimensions><<<blocks, blockSize>>>(dataVectors, centroidMemberships, membershipChangeCounter, newCentroidVectors, centroidMembershipCounts, centroidVectors, centroidVectorLength, dataVectorLength);
	cudaCheck(hipGetLastError(), CGLE_ERR);
	cudaCheck(hipDeviceSynchronize(), CDS_ERR);
}

template <int numberOfDimensions> void KMeansGPU1Solver<numberOfDimensions>::averageNewClusters()
{
	for (int i = 0; i < centroidVectorLength; i++)
		for (int j = 0; j < numberOfDimensions; j++)
			centroidVectors[i * numberOfDimensions + j] = newCentroidVectors[i * numberOfDimensions + j] / centroidMembershipCounts[i];
}

template <int numberOfDimensions> void KMeansGPU1Solver<numberOfDimensions>::clearSolver()
{
	cudaCheck(hipDeviceReset(), CDR_ERR);
}
